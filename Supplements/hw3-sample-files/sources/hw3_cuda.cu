
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_BUF 100000000
typedef unsigned int UINT;

UINT buffer[MAX_BUF];

// 核函数统一使用该命名，参数列表可自定义
__global__ void kernel() 
{
}

UINT ReadFile(const char *szFile, UINT data[])
{
	UINT len;
	FILE *fp;
	fp = fopen(szFile, "rb");
	fread(&len, sizeof(UINT), 1, fp);
	if (len > MAX_BUF)
	{
		fclose(fp);
		return 0;
	}
	fread(data, sizeof(UINT), len, fp);
	fclose(fp);
	return len;
}

void WriteFile(const char *szFile, UINT data[], UINT len)
{
	FILE *fp;
	if (len > MAX_BUF)
		return;
	fp = fopen(szFile, "wb");
	fwrite(&len, sizeof(UINT), 1, fp);
	fwrite(data, sizeof(UINT), len, fp);
	fclose(fp);
}

int main(int argc, char *argv[])
{
	UINT length;
	if (argc != 2)
		return 1;
	length = ReadFile(argv[1], buffer);
	//sorting code
	//....
	WriteFile("output.bin", buffer, length);
	return 0;
}